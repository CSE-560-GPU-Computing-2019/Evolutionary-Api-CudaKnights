#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <algorithm>  // For time()
#include <cstdlib>
#include <chrono>
#include <unistd.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>

#include "EASimpleFloat.h"

using namespace std;


float *valuesf;
float *weightf;
float maxWf;

float *matchf;
 const int fSUMFLAG=0;
 const int fKNAPSACKFLAG = 1;

const int fAVGFLAG=2;
const int fMATCHFLAG=3;
const int fINVERSESUMFLAG=4;

 const int fMAXIMIZE=-1;
 const int fMINIMIZE=1;



__global__ void setup_kernelf ( hiprandState *state, unsigned long seed )
{
    hiprand_init ( seed, 0, 0, &state[0] );
} 

__device__ float generateRandomf( hiprandState* globalState) 
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[0];
    float RANDOM = hiprand_uniform( &localState );
    globalState[0] = localState;
    return RANDOM;
}



__global__ void initializeFloat1Dpopulation(float *population,int sizeofPopulation,int sizeofChormosome,hiprandState* globalState,int division){
	int populationIndex =  blockIdx.x * blockDim.x + threadIdx.x;
	if(populationIndex<(sizeofPopulation*sizeofChormosome)){
	population[populationIndex]= (int)(generateRandomf(globalState)*2);
	//printf("CUDA %d\n",population[populationIndex]);
	
	}
__syncthreads();
}

void EAFloat::shuffle(int bias){

	std::random_shuffle(population+bias, population+populationSize);
	
}

__device__ float calculateFitnessBinary(float *chromosome,int flag,int size,float *value,float *weight,float maxLimit,float *match,int start, int end){
 	switch(flag){
 		case fKNAPSACKFLAG:
 			return getKnapsackFitness(chromosome,size,value,weight,maxLimit,start,end);
 		case fSUMFLAG:
 			return getSum(chromosome,size,start,end);
 		case fAVGFLAG:
 			return getSum(chromosome,size,start,end)/size;
 		case fMATCHFLAG:
 			return getMatch(chromosome,match,size,start,end);
 		default: return 0.0;
 	}
 
}

__device__ float getKnapsackFitness(float *chromosome, int size,float* values,float *weight,float maxW, int start,int end){
	float totalWeight=0.0;
	float value =0.0;	
	for(int i=0;i<size&&start+i<end;i++){
		float w = (chromosome[start+i])*weight[i];
		float v = (chromosome[start+i])*values[i];
		if(w+totalWeight<=maxW){
			value+=v;
			totalWeight+=w;
		}

	}
	return value;
}
__global__ void gpuCrossover(float *chromosome,hiprandState *globalState,int sizeofChromosome,int sizeofPopulation,int Bias,float prob){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int mid =(int) (generateRandomf(globalState)*sizeofChromosome);//4;// (int) (generateRandom(globalState)*(sizeofChromosome-1));
	//printf("MID: %d\n", mid);
	idx=idx*2;
	int start1,end1;

	int start2,end2;
	start1 = idx*sizeofChromosome;
	end1 = start1+sizeofChromosome;
	start2 = end1;
	end2 = start2+sizeofChromosome;
	if(end2<(sizeofChromosome*sizeofPopulation) )
	Crossover(chromosome,sizeofChromosome,start1,end1,start2,end2,mid);
	int number = (int) (generateRandomf(globalState)*100);
	if(number<(prob*100)){
		int j = (int) (generateRandomf(globalState)*((int)sizeofChromosome/4));
		for(int k=0;k<j;k++){
			int index = (int) (generateRandomf(globalState)*sizeofChromosome);
			float a = chromosome[index];// = //(chromosome[index]+1)%2;
			if(a==1){
				chromosome[index]=0;

			} 
			else{
				chromosome[index]=1;				
			}
		}
	} 

}
__global__ void calculateFitness(float *chromosome,int fitnessFlag,float *fitnessValues,int sizeofPopulation,int sizeofChromosome,float *value,float *weight,float maxLimit,float *match){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int start,end;
	start =idx*sizeofChromosome;
	end = (idx+1)*sizeofChromosome;
	fitnessValues[idx] = calculateFitnessBinary(chromosome,fitnessFlag,sizeofChromosome,value,weight,maxLimit,match,start,end);

}
void EAFloat::init()
{

//printf("Hello %d and %d\n",chromosomeSize,populationSize );
			srand(time(0));
			//hiprandState* devStates;
			auto start = chrono::steady_clock::now();
		    hipMalloc ( &devStates, sizeof( hiprandState ) );
		    auto end = chrono::steady_clock::now();
		    
		    double elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
		    totalMemoryTransferTime+=elapsed_seconds;

		    start = chrono::steady_clock::now();		    
		    setup_kernelf <<< 1, 1>>> ( devStates,unsigned(time(NULL)) );
			end = chrono::steady_clock::now();	
	
			elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
		    totalKernelTime+=elapsed_seconds;
			
			threads = dim3(chromosomeSize,1);
			blocks = dim3(populationSize,1);
			start = chrono::steady_clock::now();		
			
		   	initializeFloat1Dpopulation<<<blocks,threads>>>(Cudapopulation,chromosomeSize,popSize,devStates,4);
			end = chrono::steady_clock::now();		    
		   	elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
		   	totalKernelTime+=elapsed_seconds;
			hipDeviceSynchronize();
  
			start = chrono::steady_clock::now();		    
		   	
			for(int i=0;i<populationSize;i++){

			hipMemcpy(population[i].chromosome, Cudapopulation+(i*chromosomeSize), sizeof(float)*chromosomeSize, hipMemcpyDeviceToHost);
			//printf("Initialized: %s \t size: %d\n", population[i].chromosome,chromosomeSize);
			
			}
			end = chrono::steady_clock::now();		    
		   	
			elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
			totalMemoryTransferTime+=elapsed_seconds;

			start = chrono::steady_clock::now();		    
		   	
			fitness(0);
			end = start = chrono::steady_clock::now();		    
		   elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
			fitnessCalculationTime+=elapsed_seconds;
}

void EAFloat::setFitnessFlag(int fit,int minimax){
	fitnessFlag = fit;
	minmaxflag = minimax;
}

void EAFloat::doCrossOver(int bias){
	int s = populationSize/2;\
	/*FloatChromosome1D *pop1;
	FloatChromosome1D *pop2;
	
	pop1 = new FloatChromosome1D[populationSize];
	pop2 = new FloatChromosome1D[populationSize];

	for(int i=0;i<populationSize;i++){
	 	pop1[i].initializeChromosome(chromosomeSize);
	 	pop1[i].fitnessValue = population[i].fitnessValue;
	 	pop2[i].initializeChromosome(chromosomeSize);
	 }*/

	auto start = chrono::steady_clock::now();		    
	
	for(int i=0;i<populationSize;i++){
			hipMemcpy( Cudapopulation+(i*chromosomeSize),population[i].chromosome, sizeof(float)*chromosomeSize, hipMemcpyHostToDevice);
			//hipMemcpy( pop1[i].chromosome,Cudapopulation+(i*chromosomeSize), chromosomeSize*sizeof(int), hipMemcpyDeviceToHost);
			
	 		
	}
	auto end = chrono::steady_clock::now();		    
	
	double elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	
	totalMemoryTransferTime+=elapsed_seconds;
	bias=0;


	shuffle(bias);
	if(populationSize<256){
		threads=dim3(s,1);
		blocks=dim3(1,1);
	}
	else{
	threads = dim3(256,1);
	blocks = dim3(ceil(s/256),1);
	}

	start = chrono::steady_clock::now();		    
	
	gpuCrossover<<<blocks,threads>>>(Cudapopulation,devStates,chromosomeSize,populationSize,bias,mutationProbability);

	end = chrono::steady_clock::now();		    
	elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	totalKernelTime +=elapsed_seconds;


	start = chrono::steady_clock::now();		    
	
	for(int i=0;i<populationSize;i++){
			hipMemcpy(population[i+populationSize].chromosome, Cudapopulation+(i*chromosomeSize), sizeof(float)*chromosomeSize, hipMemcpyDeviceToHost);
		//	hipMemcpy(pop2[i].chromosome, Cudapopulation+(i*chromosomeSize), chromosomeSize*sizeof(int), hipMemcpyHostToDevice);
	}
	end = chrono::steady_clock::now();		    
	
	elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	
	totalMemoryTransferTime+=elapsed_seconds;


	start = chrono::steady_clock::now();		    
		   	
	fitness(populationSize);
	end = start = chrono::steady_clock::now();		    
	elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	fitnessCalculationTime+=elapsed_seconds;
	
	start = chrono::steady_clock::now();
	sortpop();
	end = chrono::steady_clock::now();
	elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	sortingpopulationTime+=elapsed_seconds;	
/*
	int l=0,m=0;
for(int i = 0; i < populationSize; ++i) {
			
						pop2[i].chromosome = population[i].chromosome;		
						pop2[i].fitnessValue = population[i].fitnessValue;
		}	
		int i=0;
		while(i<populationSize){
			if(pop2[l].fitnessValue<=pop1[m].fitnessValue){
				l++;
				i++;
			}
			else{
				for(int s=0;s<chromosomeSize;s++)	
				population[i].chromosome[s] = pop1[m].chromosome[s];

				i++;
				m++;
			}
		}
		delete pop1;
		delete pop2;
*/

}
void EAFloat::doMutation(int bias){


	for(int i=bias;i<populationSize;i++){
		int j=0;
		int number = rand()%100;
		
		if(number<mutationProbability*100){
			 j = rand()%chromosomeSize;
			 for(int k=0;k<j;k++){
			 	int index = rand()%chromosomeSize;

			 	//printf("MUTATING %d\n",index );
			 	population[i].chromosome[index] = ((int)population[i].chromosome[index]+1)%2;  
			 }
		}
	}
}
EAFloat::EAFloat(int sofc,int sofp,float *range)
{
	populationSize = sofp;
	popSize= populationSize*2;
	chromosomeSize = sofc;

	population = new FloatChromosome1D[popSize];
	//hipMalloc(&Cudapopulation1D, populationSize);

	indices = (int*)malloc(chromosomeSize*sizeof(int)); 
	
	for(int i=0;i<popSize;i++){
	 	population[i].initializeChromosome(chromosomeSize);
	 	//hipMalloc((void **)&Cudapopulation1D[i], chromosomeSize);
	 	//indices[i] =(rand() % chromosomeSize-1) + 2;
	 }


	 //hipMalloc((void **)&cudaIndices,populationSize*sizeof(int));
	 //hipMemcpy(cudaIndices, indices, populationSize*sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&Cudapopulation, populationSize*chromosomeSize*sizeof(float));
	hipMalloc((void**)&randomRange, chromosomeSize*sizeof(float));
	hipMemcpy(randomRange,range, chromosomeSize*sizeof(float), hipMemcpyHostToDevice);
	
}
void EAFloat::printpopulation()
{
	
	//sortpop();
	for(int i=0;i<2*populationSize;i++){
		for(int j=0;j<chromosomeSize;j++){
			printf("%f\t",population[i].chromosome[j]);

		}
		printf("\t %f\n",population[i].fitnessValue*minmaxflag);
	}
}
float EAFloat::fitness(int tag){
	float *fitnessV;
	float *realfitness;
	float avgFitness=0.0;
	realfitness = (float*) malloc(popSize*sizeof(float));
	hipMalloc((void**)&fitnessV, popSize*sizeof(float));
	if(populationSize<256){
		threads=dim3(populationSize,1);
		blocks=dim3(1,1);

	}
	else{
	threads = dim3(256,1);
	blocks = dim3(ceil(popSize/256),1);
	}
	auto start = chrono::steady_clock::now();
	calculateFitness<<<blocks,threads>>>(Cudapopulation,fitnessFlag,fitnessV,popSize,chromosomeSize,valuesf,weightf,maxWf,matchf);
	auto end = chrono::steady_clock::now();
	double elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	totalKernelTime+=elapsed_seconds;

	hipMemcpy(realfitness, fitnessV,popSize*sizeof(float), hipMemcpyDeviceToHost);
	for(int i=0;i<popSize;i++){
		population[i+tag].fitnessValue=realfitness[i]*minmaxflag;
		avgFitness+=realfitness[i];
	}
	avgFitness/=popSize;
	//printf("Average Fitness: %f\n",avgFitness);
	return avgFitness;
}

void EAFloat::sortpop(){
 std::sort(population, population + popSize,[](FloatChromosome1D const & a, FloatChromosome1D const & b) -> bool 
 			{ return (a.fitnessValue) < (b.fitnessValue); } );
}
void EAFloat::evolve()
{	int bias = 0.2 * populationSize;
	for(int i=0;i<100;i++){
		float a= fitness(0);
		if (i == 99)
			printf("Avg fitness:  %f\n", a);
		sortpop();
		doCrossOver(0);
		//printpopulation();
		//doMutation(bias);
	}

}

void EAFloat::setParamKnapSack(float *kvalues,float *kweight,int chromosomeSize,float maxWeight){
	hipMalloc((void **)&valuesf,chromosomeSize*sizeof(float));
	hipMalloc((void **)&weightf,chromosomeSize*sizeof(float));
	maxWf = maxWeight;
	hipMemcpy(valuesf,kvalues,chromosomeSize*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(weightf,kweight,chromosomeSize*sizeof(float), hipMemcpyHostToDevice);
}
void EAFloat::setMatchParameter(float *kvalues,int chromosomeSize){
	hipMalloc((void **)&matchf,chromosomeSize*sizeof(float));
	hipMemcpy(matchf,kvalues,chromosomeSize*sizeof(float), hipMemcpyHostToDevice);

}

__device__ float getMatch(float *chromosome,float *match,int size,int start,int end)
{
	float c = 0;
	for(int i = start,j=0; i < end&&j<size; i++,j++)
		if (chromosome[i] != match[j])
			c += 1;
	return c;
}

__device__ float getSum(float *chromosome,int size,int start,int end)
{
	float c = 0;
	for(int i = start; i < end; i++)
		c += chromosome[i];
	return c;
}
__device__ void Crossover(float *chromosome,int size,int start1,int end1,int start2,int end2,int  mid){

	for(int i=mid;i<size;i++){
		int c1 = start1+mid;
		int c2 = start2+mid;
		int temp = chromosome[c1];//
			//printf("temp =%d and c1 = %d and c2 = %d and ch[c2] = %d\n",temp,c1,c2,chromosome[c2]);
			
			chromosome[c1]=	chromosome[c2];
			chromosome[c2]=temp;
	}	
}

/*=======================================================================================================*/