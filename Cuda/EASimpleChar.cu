#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <ctime>
#include <algorithm>  // For time()
#include <cstdlib>
#include <chrono>
#include <unistd.h>

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <assert.h>

#include "EASimpleChar.h"

using namespace std;


float *valuesc;
float *weightc;
float maxWc;

char *matchc;
 const int cSUMFLAG=0;
 const int cKNAPSACKFLAG = 1;

const int cAVGFLAG=2;
const int cMATCHFLAG=3;
const int cINVERSESUMFLAG=4;

 const int cMAXIMIZE=-1;
 const int cMINIMIZE=1;



__global__ void setup_kernelc ( hiprandState *state, unsigned long seed )
{
    hiprand_init ( seed, 0, 0, &state[0] );
} 

__device__ float generateRandomc( hiprandState* globalState) 
{
    //int ind = threadIdx.x;
    hiprandState localState = globalState[0];
    float RANDOM = hiprand_uniform( &localState );
    globalState[0] = localState;
    return RANDOM;
}



__global__ void initializeChar1Dpopulation(char *population,int sizeofPopulation,int sizeofChormosome,hiprandState* globalState,int division){
	int populationIndex =  blockIdx.x * blockDim.x + threadIdx.x;
	if(populationIndex<(sizeofPopulation*sizeofChormosome)){
	population[populationIndex]= (char) ((int) (generateRandomc(globalState)*2)+48);
	//printf("CUDA %d\n",population[populationIndex]);
	
	}
__syncthreads();
}


__global__ void initializeChar1DRangepopulation(char *population,int sizeofPopulation,int sizeofChormosome,hiprandState* globalState,int division,char* range){
	int populationIndex =  blockIdx.x * blockDim.x + threadIdx.x;
	if(populationIndex<(sizeofPopulation*sizeofChormosome)){
	population[populationIndex]= range[(int) (generateRandomc(globalState)*sizeofChormosome)];
	//printf("CUDA %d\n",population[populationIndex]);
	
	}
__syncthreads();
}


__device__ void Crossover(char *chromosome,int size,int start1,int end1,int start2,int end2,int  mid){

	for(int i=mid;i<size;i++){
		int c1 = start1+mid;
		int c2 = start2+mid;
		int temp = chromosome[c1];//
			//printf("temp =%d and c1 = %d and c2 = %d and ch[c2] = %d\n",temp,c1,c2,chromosome[c2]);
			
			chromosome[c1]=	chromosome[c2];
			chromosome[c2]=temp;
	}	
}

void EAChar::shuffle(int bias){

	std::random_shuffle(population+bias, population+populationSize);
	
}

__device__ float calculateFitnessBinary(char *chromosome,int flag,int size,float *value,float *weight,float maxLimit,char *match,int start, int end){
 	switch(flag){
 		case cKNAPSACKFLAG:
 			return getKnapsackFitnessc(chromosome,size,value,weight,maxLimit,start,end);
 		case cSUMFLAG:
 			return getSumc(chromosome,size,start,end);
 		case cAVGFLAG:
 			return getSumc(chromosome,size,start,end)/size;
 		case cMATCHFLAG:
 			return getMatchc(chromosome,match,size,start,end);
 		default: return 0.0;
 	}
 
}

__device__ float getKnapsackFitnessc(char *chromosome, int size,float* values,float *weight,float maxW, int start,int end){
	float totalWeight=0.0;
	float value =0.0;	
	for(int i=0;i<size&&start+i<end;i++){
		float w = (chromosome[start+i]-48)*weight[i];
		float v = (chromosome[start+i]-48)*values[i];
		if(w+totalWeight<=maxW){
			value+=v;
			totalWeight+=w;
		}

	}
	return value;
}
__global__ void gpuCrossover(char *chromosome,hiprandState *globalState,int sizeofChromosome,int sizeofPopulation,int Bias,float prob,char* range){
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	int mid =(int) (generateRandomc(globalState)*sizeofChromosome);//4;// (int) (generateRandom(globalState)*(sizeofChromosome-1));
	//printf("MID: %d\n", mid);
	idx=idx*2;
	int start1,end1;

	int start2,end2;
	start1 = idx*sizeofChromosome;
	end1 = start1+sizeofChromosome;
	start2 = end1;
	end2 = start2+sizeofChromosome;
	if(end2<(sizeofChromosome*sizeofPopulation) )
	Crossover(chromosome,sizeofChromosome,start1,end1,start2,end2,mid);
	int number = (int) (generateRandomc(globalState)*100);
	if(number<(prob*100)){
		int j = (int) (generateRandomc(globalState)*((int)sizeofChromosome/4));
		for(int k=0;k<j;k++){
			int index = (int) (generateRandomc(globalState)*sizeofChromosome);
			chromosome[index] = range[(int) (generateRandomc(globalState)*sizeofChromosome)];// = //(chromosome[index]+1)%2;
			// if(a=='1'){
			// 	chromosome[index]='0';

			// } 
			// else{
			// 	chromosome[index]='1';				
			// }
		}
	} 

}
__global__ void calculateFitness(char *chromosome,int fitnessFlag,float *fitnessValues,int sizeofPopulation,int sizeofChromosome,float *value,float *weight,float maxLimit,char *match){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int start,end;
	start =idx*sizeofChromosome;
	end = (idx+1)*sizeofChromosome;
	fitnessValues[idx] = calculateFitnessBinary(chromosome,fitnessFlag,sizeofChromosome,value,weight,maxLimit,match,start,end);

}
void EAChar::init()
{

//printf("Hello %d and %d\n",chromosomeSize,populationSize );
			srand(time(0));
			//hiprandState* devStates;
			auto start = chrono::steady_clock::now();
		    hipMalloc ( &devStates, sizeof( hiprandState ) );
		    auto end = chrono::steady_clock::now();
		    
		    double elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
		    totalMemoryTransferTime+=elapsed_seconds;

		    start = chrono::steady_clock::now();		    
		    setup_kernelc <<< 1, 1>>> ( devStates,unsigned(time(NULL)) );
			end = chrono::steady_clock::now();	
	
			elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
		    totalKernelTime+=elapsed_seconds;
			
			threads = dim3(chromosomeSize,1);
			blocks = dim3(populationSize,1);
			start = chrono::steady_clock::now();		
			
		   	initializeChar1DRangepopulation<<<blocks,threads>>>(Cudapopulation,chromosomeSize,popSize,devStates,4,randomRange);
			end = chrono::steady_clock::now();		    
		   	elapsed_seconds = std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
		   	totalKernelTime+=elapsed_seconds;
			hipDeviceSynchronize();
  
			start = chrono::steady_clock::now();		    
		   	
			for(int i=0;i<populationSize;i++){

			hipMemcpy(population[i].chromosome, Cudapopulation+(i*chromosomeSize), sizeof(char)*chromosomeSize, hipMemcpyDeviceToHost);
			//printf("Initialized: %s \t size: %d\n", population[i].chromosome,chromosomeSize);
			
			}
			end = chrono::steady_clock::now();		    
		   	
			elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
			totalMemoryTransferTime+=elapsed_seconds;

			start = chrono::steady_clock::now();		    
		   	
			fitness(0);
			end = start = chrono::steady_clock::now();		    
		   elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
			fitnessCalculationTime+=elapsed_seconds;
}

void EAChar::setFitnessFlag(int fit,int minimax){
	fitnessFlag = fit;
	minmaxflag = minimax;
}

void EAChar::doCrossOver(int bias){
	int s = populationSize/2;\
	/*CharChromosome1D *pop1;
	CharChromosome1D *pop2;
	
	pop1 = new CharChromosome1D[populationSize];
	pop2 = new CharChromosome1D[populationSize];

	for(int i=0;i<populationSize;i++){
	 	pop1[i].initializeChromosome(chromosomeSize);
	 	pop1[i].fitnessValue = population[i].fitnessValue;
	 	pop2[i].initializeChromosome(chromosomeSize);
	 }*/

	auto start = chrono::steady_clock::now();		    
	
	for(int i=0;i<populationSize;i++){
			hipMemcpy( Cudapopulation+(i*chromosomeSize),population[i].chromosome, sizeof(char)*chromosomeSize, hipMemcpyHostToDevice);
			//hipMemcpy( pop1[i].chromosome,Cudapopulation+(i*chromosomeSize), chromosomeSize*sizeof(int), hipMemcpyDeviceToHost);
			
	 		
	}
	auto end = chrono::steady_clock::now();		    
	
	double elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	
	totalMemoryTransferTime+=elapsed_seconds;
	bias=0;


	shuffle(bias);
	if(populationSize<256){
		threads=dim3(s,1);
		blocks=dim3(1,1);
	}
	else{
	threads = dim3(256,1);
	blocks = dim3(ceil(s/256),1);
	}

	start = chrono::steady_clock::now();		    
	
	gpuCrossover<<<blocks,threads>>>(Cudapopulation,devStates,chromosomeSize,populationSize,bias,mutationProbability,randomRange);

	end = chrono::steady_clock::now();		    
	elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	totalKernelTime +=elapsed_seconds;


	start = chrono::steady_clock::now();		    
	
	for(int i=0;i<populationSize;i++){
			hipMemcpy(population[i+populationSize].chromosome, Cudapopulation+(i*chromosomeSize), sizeof(char)*chromosomeSize, hipMemcpyDeviceToHost);
		//	hipMemcpy(pop2[i].chromosome, Cudapopulation+(i*chromosomeSize), chromosomeSize*sizeof(int), hipMemcpyHostToDevice);
	}
	end = chrono::steady_clock::now();		    
	
	elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	
	totalMemoryTransferTime+=elapsed_seconds;


	start = chrono::steady_clock::now();		    
		   	
	fitness(populationSize);
	end = start = chrono::steady_clock::now();		    
	elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	fitnessCalculationTime+=elapsed_seconds;
	
	start = chrono::steady_clock::now();
	sortpop();
	end = chrono::steady_clock::now();
	elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	sortingpopulationTime+=elapsed_seconds;	
/*
	int l=0,m=0;
for(int i = 0; i < populationSize; ++i) {
			
						pop2[i].chromosome = population[i].chromosome;		
						pop2[i].fitnessValue = population[i].fitnessValue;
		}	
		int i=0;
		while(i<populationSize){
			if(pop2[l].fitnessValue<=pop1[m].fitnessValue){
				l++;
				i++;
			}
			else{
				for(int s=0;s<chromosomeSize;s++)	
				population[i].chromosome[s] = pop1[m].chromosome[s];

				i++;
				m++;
			}
		}
		delete pop1;
		delete pop2;
*/

}
void EAChar::doMutation(int bias){


	for(int i=bias;i<populationSize;i++){
		int j=0;
		int number = rand()%100;
		
		if(number<mutationProbability*100){
			 j = rand()%chromosomeSize;
			 for(int k=0;k<j;k++){
			 	int index = rand()%chromosomeSize;

			 	//printf("MUTATING %d\n",index );
			 	population[i].chromosome[index] = (char) (((int)population[i].chromosome[index]+1-48)%2 + 48);  
			 }
		}
	}
}
EAChar::EAChar(int sofc,int sofp,char *range)
{
	populationSize = sofp;
	popSize= populationSize*2;
	chromosomeSize = sofc;

	population = new CharChromosome1D[popSize];
	//hipMalloc(&Cudapopulation1D, populationSize);
	
	for(int i=0;i<popSize;i++){
	 	population[i].initializeChromosome(chromosomeSize);
	 	//hipMalloc((void **)&Cudapopulation1D[i], chromosomeSize);
	 	//indices[i] =(rand() % chromosomeSize-1) + 2;
	 }


	 //hipMalloc((void **)&cudaIndices,populationSize*sizeof(int));
	 //hipMemcpy(cudaIndices, indices, populationSize*sizeof(int), hipMemcpyHostToDevice);
	hipMalloc((void**)&Cudapopulation, populationSize*chromosomeSize*sizeof(char));
	hipMalloc((void**)&randomRange, chromosomeSize*sizeof(char));
	hipMemcpy(randomRange,range, chromosomeSize*sizeof(char), hipMemcpyHostToDevice);
	
}
void EAChar::printpopulation()
{
	
	//sortpop();
	for(int i=0;i<2*populationSize;i++){
		for(int j=0;j<chromosomeSize;j++){
			printf("%c",population[i].chromosome[j]);

		}
		printf("\t %f\n",population[i].fitnessValue*minmaxflag);
	}
}
float EAChar::fitness(int tag){
	float *fitnessV;
	float *realfitness;
	float avgFitness=0.0;
	realfitness = (float*) malloc(popSize*sizeof(float));
	hipMalloc((void**)&fitnessV, popSize*sizeof(float));
	if(populationSize<256){
		threads=dim3(populationSize,1);
		blocks=dim3(1,1);

	}
	else{
	threads = dim3(256,1);
	blocks = dim3(ceil(popSize/256),1);
	}
	auto start = chrono::steady_clock::now();
	calculateFitness<<<blocks,threads>>>(Cudapopulation,fitnessFlag,fitnessV,popSize,chromosomeSize,valuesc,weightc,maxWc,matchc);
	auto end = chrono::steady_clock::now();
	double elapsed_seconds =  std::chrono::duration_cast<std::chrono::duration<double> >(end-start).count();
	totalKernelTime+=elapsed_seconds;

	hipMemcpy(realfitness, fitnessV,popSize*sizeof(float), hipMemcpyDeviceToHost);
	for(int i=0;i<popSize;i++){
		population[i+tag].fitnessValue=realfitness[i]*minmaxflag;
		avgFitness+=realfitness[i];
	}
	avgFitness/=popSize;
	//printf("Average Fitness: %f\n",avgFitness);
	return avgFitness;
}

void EAChar::sortpop(){
 std::sort(population, population + popSize,[](CharChromosome1D const & a, CharChromosome1D const & b) -> bool 
 			{ return (a.fitnessValue) < (b.fitnessValue); } );
}
void EAChar::evolve()
{	int bias = 0.2 * populationSize;
	for(int i=0;i<100;i++){
		float a= fitness(0);
		if (i == 99)
			printf("Avg fitness:  %f\n", a);
		sortpop();
		doCrossOver(0);
		//printpopulation();
		//doMutation(bias);
	}

}

void EAChar::setParamKnapSack(float *kvalues,float *kweight,int chromosomeSize,float maxWeight){
	hipMalloc((void **)&valuesc,chromosomeSize*sizeof(float));
	hipMalloc((void **)&weightc,chromosomeSize*sizeof(float));
	maxWc = maxWeight;
	hipMemcpy(valuesc,kvalues,chromosomeSize*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(weightc,kweight,chromosomeSize*sizeof(float), hipMemcpyHostToDevice);
}
void EAChar::setMatchParameter(char *kvalues,int chromosomeSize){
	hipMalloc((void **)&matchc,chromosomeSize*sizeof(char));
	hipMemcpy(matchc,kvalues,chromosomeSize*sizeof(char), hipMemcpyHostToDevice);

}

__device__ float getMatchc(char *chromosome,char *match,int size,int start,int end)
{
	float c = 0;
	for(int i = start,j=0; i < end&&j<size; i++,j++)
		if (chromosome[i] != match[j])
			c += 1;
	return c;
}

__device__ float getSumc(char *chromosome,int size,int start,int end)
{
	float c = 0;
	for(int i = start; i < end; i++)
		c += (int)chromosome[i]-48;
	return c;
}

/*=======================================================================================================*/